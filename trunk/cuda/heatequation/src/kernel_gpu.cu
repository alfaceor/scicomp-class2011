#include "hip/hip_runtime.h"
#include "kernel_gpu.cuH"

/**
Generate the new u in the next time step using blocks
*/
__global__ void new_time_blocks(float *u, float *u_new,float c) {
    // XXX: Because the boundary conditions, this must started in 1
    // u_[1], that is the reason why i added +1
    int tid = blockIdx.x;  // this thread handles the data at its thread id
	// The array is a matrix (N+1)*(N)
	// Just chain the values of u in the interval 1,rowdim

    if (tid < N){
        // If tid belows to the boundary don't make anything(fixed boundary cond)
        if(tid<DIM || tid > N-1-DIM){           // Horizontal boundary
            u_new[tid]=u[tid];
        }else if(tid%DIM==0 || (tid+1)%DIM==0){ //Vertical Boundary
            u_new[tid]=u[tid];
        }else{  // make a time step
            u_new[tid] = u[tid]+c*(u[tid+1]+u[tid-1]+u[tid-DIM]+u[tid+DIM]-4*u[tid]);
        }
        
    }
}

/**
Generate the new u in the next time step using threads
*/

__global__ void new_time_threads(float *u, float *u_new,float c) {
    int tid = blockIdx.x;    // this thread handles the data at its thread id
    if (tid < N)
        u_new[tid] = u[tid+1]+u[tid-1]-4*u[tid] ;
        //u_new[tid] = u[tid+1]+u[tid-1]+u[tid-dim]+u[tid+dim]-4*u[tid] ;
}

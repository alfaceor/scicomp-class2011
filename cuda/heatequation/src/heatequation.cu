/**
Necesito las siguientes matrices u_new, u_old, m
*/

#include "kernel_gpu.cuH"
#include <stdio.h>
#include <stdlib.h>

int main( int argc, char *argv ) {

    if(argc!=2){
       printf("usage: heatequation <number_of_steps*2>\n");
       return 1;
    }
    int steps_2=atoi(argv[1]);  // time 2 time steps
    int steps=1000;
    float u[N], u_new[N];   // N must be a perfect square number
    float *dev_u, *dev_u_new;
    float c=0.001;

    struct timeval tempo1, tempo2;
    
    long elapsed_utime;    /* elapsed time in microseconds */
    long elapsed_mtime;    /* elapsed time in milliseconds */
    long elapsed_seconds;  /* diff between seconds counter */
    long elapsed_useconds; /* diff between microseconds counter */



    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_u, N * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_u_new, N * sizeof(float) ) );

    // fill the array 'u' on the CPU
    // initial condition
    for (int i=0; i<N; i++){
        u[i]=0;
    }
    for (int i=0; i<DIM; i++) {
    // Horizontal boundary conditions
        u[i]       = i;
        u[i*DIM]   = i;
        u[N-1-i]     = i;
        u[N-1-i*DIM] = i;
    }

/*
//    printf("****Print u_old matrix****\n");
//    // display the results
//    for (int i=0; i<DIM; i++) {
//        for (int j=0; j<DIM; j++) {
//            printf( "%f\t", u[i*DIM+j] );
//        }
//        printf("----\n");
//    }
*/
    // copy the array 'u' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_u, u, N * sizeof(float),
                              hipMemcpyHostToDevice ) );

//    printf("Press 'Enter' to show the elapsed time: ");
    gettimeofday(&tempo1, NULL);
    for (int i=0;i<steps;i++){
        if (i%2==0)
            new_time_blocks<<<N,1>>>( dev_u, dev_u_new, c);
        else
            new_time_blocks<<<N,1>>>( dev_u_new, dev_u, c);
    }

//
//    gettimeofday(&tempo2, NULL);
//    printf("tempo2.tv_sec = %ld, tempo2_tv_usec = %ld\n",
//            tempo2.tv_sec, tempo2.tv_usec);
//    printf("tempo1.tv_sec = %ld, tempo1_tv_usec = %ld\n",
//            tempo1.tv_sec, tempo1.tv_usec);
//    elapsed_seconds  = tempo2.tv_sec  - tempo1.tv_sec;
//    elapsed_useconds = tempo2.tv_usec - tempo1.tv_usec;
//
//    printf("Elapsed time = %ld seconds + %ld microseconds\n",
//            elapsed_seconds, elapsed_useconds);
//
//    elapsed_utime = (elapsed_seconds) * 1000000 + elapsed_useconds;
//    elapsed_mtime = ((elapsed_seconds) * 1000 + elapsed_useconds/1000.0) + 0.5;
//
//    printf("Elapsed time = %ld microseconds\n", elapsed_utime);
//    printf("Elapsed time = %ld milliseconds\n", elapsed_mtime);
//



    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( u_new, dev_u_new, N * sizeof(float),
                              hipMemcpyDeviceToHost ) );
    
//    printf("****Print u_new matrix****\n");
    // display the results
    for (int i=0; i<DIM; i++) {
        for (int j=0; j<DIM; j++) {
            printf( "%f\t", u_new[i*DIM+j] );
        }
        printf("\n");
    }

    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( dev_u ) );
    HANDLE_ERROR( hipFree( dev_u_new ) );

    return 0;
}
